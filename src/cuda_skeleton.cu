#include "hip/hip_runtime.h"
#include "decom.h"

using namespace std;

__global__ void test_Kernel(int* d_num_v1)
{
    int threadID = threadIdx.x;
    cout << "threadID: " << threadID <<  "d_num_v1: " << *d_num_v1 << "\n";
}

void cuda_query(string dir, int num_blocks_per_grid, int num_threads_per_block, int* queryAns) {
	BiGraph h_g(dir);
	lrIndexBasic(h_g);    
    vector<vector<lrval_index_block*>> h_lrval_index_u; vector<vector<lrval_index_block*>> h_lrval_index_v;
    build_lrval_index(h_g, h_lrval_index_u, h_lrval_index_v);

    size_t size_num_v1 = sizeof(int);
    size_t size_num_v2 = sizeof(int);
    size_t size_h_lrval_index_u = sizeof(lrval_index_block*)*h_lrval_index_u[0].size()*h_lrval_index_u.size();
    size_t size_h_lrval_index_v = sizeof(lrval_index_block*)*h_lrval_index_v[0].size()*h_lrval_index_v.size();
    
    int *d_num_v1;
    int *d_num_v2;
    vector<vector<lrval_index_block*>> *d_lrval_index_u;
    vector<vector<lrval_index_block*>> *d_lrval_index_v;
    
    hipMalloc((void**)&d_num_v1,size_num_v1);
    hipMalloc((void**)&d_num_v2,size_num_v2);
    test_Kernel<<<num_blocks_per_grid,num_threads_per_block>>>(d_num_v1);
    exit(0);
    hipMalloc(&d_lrval_index_u,size_h_lrval_index_u);
    hipMalloc(&d_lrval_index_v,size_h_lrval_index_v);
    hipMemcpy(d_num_v1,&h_g.num_v1,size_num_v1,hipMemcpyHostToDevice);
    hipMemcpy(d_num_v2,&h_g.num_v2,size_num_v2,hipMemcpyHostToDevice);
    hipMemcpy(d_lrval_index_u,&h_lrval_index_u,size_h_lrval_index_u,hipMemcpyHostToDevice);
    hipMemcpy(d_lrval_index_v,&h_lrval_index_v,size_h_lrval_index_v,hipMemcpyHostToDevice);
    
    exit(0);
    // vector<bool> left; vector<bool> right;
    // // all the vertices in query result are set as true
    // vector<vector<int>> queryStream;
    // queryStream.resize(Q_MAX);
    // int n_query = 0;
    // loadQuery(argv[2], queryStream, n_query);
    // queryStream.resize(n_query);
    // int queryAns[n_query*3];    




}
