#include "hip/hip_runtime.h"
#include "decom.h"

using namespace std;
const int Q_MAX = 10000000;
__global__ void Kernel(int* d_lrval_index_u_size,int* d_queryStream,int* d_queryAns,int* d_n_query,int* d_lrval_index_u_length)
{
    const int tid = blockDim.x*blockIdx.x + threadIdx.x;
    const int nthread = blockDim.x*gridDim.x; 

    for(int i = tid;i<*d_n_query; i+= nthread){
        int flag = 0;
        int lval = d_queryStream[i*2];
        int rval = d_queryStream[i*2+1];
        if ((*d_lrval_index_u_length<= lval) || (d_lrval_index_u_size[lval] <= rval)){
		    flag = 0;
        } else {
            flag = 1;
        }

        d_queryAns[i*3] = lval;
        d_queryAns[i*3+1] = rval;
        d_queryAns[i*3+2] = flag;
    }
    
}


void cuda_query(string dir, int num_blocks_per_grid, int num_threads_per_block, int* queryAns) {
	BiGraph h_g(dir);
	lrIndexBasic(h_g);    
    vector<vector<lrval_index_block*>> h_lrval_index_u; vector<vector<lrval_index_block*>> h_lrval_index_v;
    build_lrval_index(h_g, h_lrval_index_u, h_lrval_index_v);

    int *h_lrval_index_u_size,*d_lrval_index_u_size;
    int h_lrval_index_u_length = (int)h_lrval_index_u.size();
    int *d_lrval_index_u_length;
    
    hipMalloc((void**)&d_lrval_index_u_length,sizeof(int));
    hipMemcpy(d_lrval_index_u_length,&h_lrval_index_u_length,sizeof(int),hipMemcpyHostToDevice);

    size_t size_h_lrval_index_u_size = sizeof(h_lrval_index_u.size()) * h_lrval_index_u.size();
    h_lrval_index_u_size = (int*)malloc(size_h_lrval_index_u_size);
    
    for (int i = 0;i<h_lrval_index_u.size();i++){
        h_lrval_index_u_size[i] = h_lrval_index_u[i].size();
    }
    
    hipMalloc((void**)&d_lrval_index_u_size,size_h_lrval_index_u_size);
    hipMemcpy(d_lrval_index_u_size,h_lrval_index_u_size,size_h_lrval_index_u_size,hipMemcpyHostToDevice);

    vector<vector<int>> queryStream;
    queryStream.resize(Q_MAX);
    int n_query = sizeof(queryAns)/sizeof(queryAns[0]);
    int *d_n_query;

    loadQuery(dir, queryStream);
    queryStream.resize(n_query);
    hipMalloc((void**)&d_n_query,sizeof(int));
    hipMemcpy(d_n_query,&n_query,sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(&n_query,d_n_query,sizeof(int),hipMemcpyDeviceToHost);
    
    
    int *h_queryStream,*d_queryStream;
    size_t size_h_query = sizeof(queryStream[0][0]) * n_query * 2;
    h_queryStream = (int*)malloc(size_h_query);
    for (int i = 0;i<n_query;i++){
        h_queryStream[i*2] = queryStream[i][0];
        h_queryStream[i*2+1] = queryStream[i][1];
    }

    hipMalloc((void**)&d_queryStream,size_h_query);
    hipMemcpy(d_queryStream,h_queryStream,size_h_query,hipMemcpyHostToDevice);

    
    int *d_queryAns;
    size_t size_h_queryAns = sizeof(int)*n_query*3;

    hipMalloc((void**)&d_queryAns,size_h_queryAns);
    hipMemcpy(d_queryAns,queryAns,size_h_queryAns,hipMemcpyHostToDevice);
    Kernel<<<num_blocks_per_grid,num_threads_per_block>>>(d_lrval_index_u_size,d_queryStream,d_queryAns,d_n_query,d_lrval_index_u_length);

    hipMemcpy(queryAns,d_queryAns,size_h_queryAns,hipMemcpyDeviceToHost);



    






}
