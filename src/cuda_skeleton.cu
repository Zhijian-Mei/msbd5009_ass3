#include "hip/hip_runtime.h"
#include "decom.h"

using namespace std;

__global__ void test_Kernel(int* C,int* num_v1,int* num_v2)
{
    int threadID = threadIdx.x;
    
}

void cuda_query(string dir, int num_blocks_per_grid, int num_threads_per_block, int* queryAns) {
	BiGraph h_g(dir);
	lrIndexBasic(h_g);    
    vector<vector<lrval_index_block*>> h_lrval_index_u; vector<vector<lrval_index_block*>> h_lrval_index_v;
    build_lrval_index(h_g, h_lrval_index_u, h_lrval_index_v);
    int num_h_lrval_index_u = 0;
    int num_h_lrval_index_v = 0;
    for (int i = 0;i<h_lrval_index_u.size();i++){
        num_h_lrval_index_u = num_h_lrval_index_u + h_lrval_index_u[i].size();
    }
    for (int i = 0;i<h_lrval_index_v.size();i++){
        num_h_lrval_index_v = num_h_lrval_index_v + h_lrval_index_v[i].size();
    }
    cout<<num_h_lrval_index_u<<" "<< num_h_lrval_index_v<<"\n";
    exit(0);
    size_t size = 2 * sizeof(int);
    size_t size_num_v1 = sizeof(int);
    size_t size_num_v2 = sizeof(int);
    size_t size_h_lrval_index_u = sizeof(lrval_index_block*)*h_lrval_index_u[0].size()*h_lrval_index_u.size();
    size_t size_h_lrval_index_v = sizeof(lrval_index_block*)*h_lrval_index_v[0].size()*h_lrval_index_v.size();
    cout<<size_h_lrval_index_u<<" "<<size_num_v1<<"\n";
    exit(0);
    // int *h_c,*d_c;
    // h_c = (int*)malloc(size);
    // hipMalloc((void**)&d_c,size);
    // hipMemcpy(d_c,h_c,size,hipMemcpyHostToDevice);
    // test_Kernel<<<num_blocks_per_grid,num_threads_per_block>>>(d_c);
    // hipMemcpy(h_c,d_c,size,hipMemcpyDeviceToHost);
    // cout<<h_c[0]<<" "<<h_c[1]<<"\n";
    // exit(0);



    int *d_num_v1;
    int *d_num_v2;
    
    hipMalloc((void**)&d_num_v1,size_num_v1);
    hipMalloc((void**)&d_num_v2,size_num_v2);
    hipMemcpy(d_num_v1,&h_g.num_v1,size_num_v1,hipMemcpyHostToDevice);
    hipMemcpy(d_num_v2,&h_g.num_v2,size_num_v2,hipMemcpyHostToDevice);

    // int *h_c,*d_c;
    // h_c = (int*)malloc(size);
    // hipMalloc((void**)&d_c,size);
    // hipMemcpy(d_c,h_c,size,hipMemcpyHostToDevice);
    // test_Kernel<<<num_blocks_per_grid,num_threads_per_block>>>(d_c,d_num_v1,d_num_v2);
    // hipMemcpy(h_c,d_c,size,hipMemcpyDeviceToHost);
    // exit(0);

    vector<vector<lrval_index_block*>> *d_lrval_index_u;
    vector<vector<lrval_index_block*>> *d_lrval_index_v;
    hipMalloc(&d_lrval_index_u,size_h_lrval_index_u);
    hipMalloc(&d_lrval_index_v,size_h_lrval_index_v);
    hipMemcpy(d_lrval_index_u,&h_lrval_index_u,size_h_lrval_index_u,hipMemcpyHostToDevice);
    hipMemcpy(d_lrval_index_v,&h_lrval_index_v,size_h_lrval_index_v,hipMemcpyHostToDevice);
    
    exit(0);
    // vector<bool> left; vector<bool> right;
    // // all the vertices in query result are set as true
    // vector<vector<int>> queryStream;
    // queryStream.resize(Q_MAX);
    // int n_query = 0;
    // loadQuery(argv[2], queryStream, n_query);
    // queryStream.resize(n_query);
    // int queryAns[n_query*3];    




}
