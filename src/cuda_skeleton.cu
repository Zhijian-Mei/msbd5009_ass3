#include "hip/hip_runtime.h"
#include "decom.h"

using namespace std;

__global__ void test_Kernel(int* d_lrval_index_u_size)
{
    int threadID = threadIdx.x;
    printf(d_lrval_index_u_size[threadID]);
}

void cuda_query(string dir, int num_blocks_per_grid, int num_threads_per_block, int* queryAns) {
	BiGraph h_g(dir);
	lrIndexBasic(h_g);    
    vector<vector<lrval_index_block*>> h_lrval_index_u; vector<vector<lrval_index_block*>> h_lrval_index_v;
    build_lrval_index(h_g, h_lrval_index_u, h_lrval_index_v);
    int num_h_lrval_index_u = 0;
    int num_h_lrval_index_v = 0;

    // size_t size = 2 * sizeof(int);
    size_t size_num_v1 = sizeof(int);
    size_t size_num_v2 = sizeof(int);




    int *d_num_v1;
    int *d_num_v2;
    
    hipMalloc((void**)&d_num_v1,size_num_v1);
    hipMalloc((void**)&d_num_v2,size_num_v2);
    hipMemcpy(d_num_v1,&h_g.num_v1,size_num_v1,hipMemcpyHostToDevice);
    hipMemcpy(d_num_v2,&h_g.num_v2,size_num_v2,hipMemcpyHostToDevice);

    // int *h_c,*d_c;
    // h_c = (int*)malloc(size);
    // hipMalloc((void**)&d_c,size);
    // hipMemcpy(d_c,h_c,size,hipMemcpyHostToDevice);
    // test_Kernel<<<num_blocks_per_grid,num_threads_per_block>>>(d_c,d_num_v1,d_num_v2);
    // hipMemcpy(h_c,d_c,size,hipMemcpyDeviceToHost);
    // exit(0);
    int h_lrval_index_u_size[h_lrval_index_u.size()];
    int *d_lrval_index_u_size;
    size_t size_h_lrval_index_u_size = sizeof(int) * h_lrval_index_u.size();
    for (int i = 0;i<h_lrval_index_u.size();i++){
        h_lrval_index_u_size[i] = h_lrval_index_u[i].size();
    }
    hipMalloc((void**)&d_lrval_index_u_size,size_h_lrval_index_u_size);
    hipMemcpy(d_lrval_index_u_size,&h_lrval_index_u_size,size_d_lrval_index_u_size,hipMemcpyHostToDevice);
    test_Kernel<<<num_blocks_per_grid,num_threads_per_block>>>(d_lrval_index_u_size);
    
    exit(0);
    // vector<bool> left; vector<bool> right;
    // // all the vertices in query result are set as true
    // vector<vector<int>> queryStream;
    // queryStream.resize(Q_MAX);
    // int n_query = 0;
    // loadQuery(argv[2], queryStream, n_query);
    // queryStream.resize(n_query);
    // int queryAns[n_query*3];    




}
