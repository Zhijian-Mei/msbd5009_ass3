#include "hip/hip_runtime.h"
#include "decom.h"

using namespace std;
const int Q_MAX = 10000000;
__global__ void Kernel(int* d_lrval_index_u_size,int* d_queryStream,int* d_queryAns,int* d_n_query,int* d_lrval_index_u_length)
{
    const int tid = blockDim.x*blockIdx.x + threadIdx.x;
    const int nthread = blockDim.x*gridDim.x; 

    for(int i = tid;i<*d_n_query; i+= nthread){
        int flag = 0;
        int lval = d_queryStream[i*2];
        int rval = d_queryStream[i*2+1];
        if ((*d_lrval_index_u_length<= lval) || (d_lrval_index_u_size[i] <= rval)){
		    flag = 0;
        } else {
            flag = 1;
        }

        d_queryAns[i*3] = lval;
        d_queryAns[i*3+1] = rval;
        d_queryAns[i*3+2] = flag;
    }
    
}
__global__ void test(int* d_lrval_index_u_size,int* d_queryStream,int* d_queryAns,int* d_n_query,int* d_c,int* d_lrval_index_u_length)
{
    const int tid = blockDim.x*blockIdx.x + threadIdx.x;
    const int nthread = blockDim.x*gridDim.x; 

    for(int i = tid;i<*d_n_query; i+= nthread){
        int flag = 0;
        int lval = d_queryStream[i*2];
        int rval = d_queryStream[i*2+1];
        if ((*d_lrval_index_u_length<= lval) || (d_lrval_index_u_size[i] <= rval)){
		    flag = 0;
        } else {
            flag = 1;
        }

        d_queryAns[i*3] = lval;
        d_queryAns[i*3+1] = rval;
        d_queryAns[i*3+2] = flag;
    }
    
}

void loadQuery(string dir, std::vector<std::vector<int>>& queryStream,int &line)
{
	int r, lval, rval;
	string queryFile = dir + "querystream.txt";
	FILE * queryVec = fopen(queryFile.c_str(), "r");
	line = 0;
	while ((r = fscanf(queryVec, "%d %d", &lval, &rval)) != EOF)
	{
		if (r != 2)
		{
			fprintf(stderr, "Bad file format: u v incorrect\n");
			exit(1);
		}
		queryStream[line].resize(2);
		queryStream[line][0] = lval;
		queryStream[line][1] = rval;
		line++;
	}
	// cout<<"line: " << line;

	fclose(queryVec);
}

void cuda_query(string dir, int num_blocks_per_grid, int num_threads_per_block, int* queryAns) {
	BiGraph h_g(dir);
	lrIndexBasic(h_g);    
    vector<vector<lrval_index_block*>> h_lrval_index_u; vector<vector<lrval_index_block*>> h_lrval_index_v;
    build_lrval_index(h_g, h_lrval_index_u, h_lrval_index_v);

    // size_t size = 2 * sizeof(int);
    // size_t size_num_v1 = sizeof(int);
    // size_t size_num_v2 = sizeof(int);




    // int *d_num_v1;
    // int *d_num_v2;
    
    // hipMalloc((void**)&d_num_v1,size_num_v1);
    // hipMalloc((void**)&d_num_v2,size_num_v2);
    // hipMemcpy(d_num_v1,&h_g.num_v1,size_num_v1,hipMemcpyHostToDevice);
    // hipMemcpy(d_num_v2,&h_g.num_v2,size_num_v2,hipMemcpyHostToDevice);


    int *h_lrval_index_u_size,*d_lrval_index_u_size,*d_lrval_index_u_length;
    int h_lrval_index_u_length = h_lrval_index_u.size();
    

    hipMalloc((void**)&d_lrval_index_u_length,sizeof(h_lrval_index_u.size()));
    hipMemcpy(d_lrval_index_u_length,&h_lrval_index_u_length,sizeof(h_lrval_index_u.size()),hipMemcpyHostToDevice);
    hipMemcpy(&h_lrval_index_u_length,d_lrval_index_u_length,,sizeof(h_lrval_index_u.size()),hipMemcpyDeviceToHost);
    cout<<h_lrval_index_u.size()<<"\n";
    cout<<h_lrval_index_u_length<<"\n";
    
    size_t size_h_lrval_index_u_size = sizeof(h_lrval_index_u.size()) * h_lrval_index_u.size();
    h_lrval_index_u_size = (int*)malloc(size_h_lrval_index_u_size);
    
    for (int i = 0;i<h_lrval_index_u.size();i++){
        h_lrval_index_u_size[i] = h_lrval_index_u[i].size();
    }
    
    hipMalloc((void**)&d_lrval_index_u_size,size_h_lrval_index_u_size);
    hipMemcpy(d_lrval_index_u_size,h_lrval_index_u_size,size_h_lrval_index_u_size,hipMemcpyHostToDevice);
    
    // test_Kernel<<<num_blocks_per_grid,num_threads_per_block>>>(d_lrval_index_u_size);

    vector<vector<int>> queryStream;
    queryStream.resize(Q_MAX);
    int n_query = 0;
    int *d_n_query;

    loadQuery(dir, queryStream,n_query);
    queryStream.resize(n_query);
    hipMalloc((void**)&d_n_query,sizeof(int));
    hipMemcpy(d_n_query,&n_query,sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(&n_query,d_n_query,sizeof(int),hipMemcpyDeviceToHost);
    cout<<n_query<<"\n";
    
    
    int *h_queryStream,*d_queryStream;
    size_t size_h_query = sizeof(queryStream[0][0]) * n_query * 2;
    h_queryStream = (int*)malloc(size_h_query);
    for (int i = 0;i<n_query;i++){
        h_queryStream[i*2] = queryStream[i][0];
        h_queryStream[i*2+1] = queryStream[i][1];
    }

    hipMalloc((void**)&d_queryStream,size_h_query);
    hipMemcpy(d_queryStream,h_queryStream,size_h_query,hipMemcpyHostToDevice);
    // test_Kernel<<<num_blocks_per_grid,num_threads_per_block>>>(d_lrval_index_u_size,d_queryStream);
    // int *h_c,*d_c;
    // h_c = (int*)malloc(sizeof(int)*100);
    // hipMalloc((void**)&d_c,sizeof(int)*100);
    // hipMemcpy(d_c,h_c,sizeof(int)*100,hipMemcpyHostToDevice);
    // test<<<num_blocks_per_grid,num_threads_per_block>>>(d_lrval_index_u_size,d_queryStream,d_n_query,d_c);
    // hipMemcpy(h_c,d_c,sizeof(int)*100,hipMemcpyDeviceToHost);
    // cout<<h_c[0]<<" "<<h_c[1]<<"\n";
    // exit(0);


    
    int *d_queryAns;
    size_t size_h_queryAns = sizeof(int)*n_query*3;

    hipMalloc((void**)&d_queryAns,size_h_queryAns);
    hipMemcpy(d_queryAns,queryAns,size_h_queryAns,hipMemcpyHostToDevice);
    Kernel<<<num_blocks_per_grid,num_threads_per_block>>>(d_lrval_index_u_size,d_queryStream,d_queryAns,d_n_query,d_lrval_index_u_length);

    hipMemcpy(queryAns,d_queryAns,size_h_queryAns,hipMemcpyDeviceToHost);



    






}
