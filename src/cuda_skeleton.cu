#include "hip/hip_runtime.h"
#include "decom.h"

using namespace std;
const int Q_MAX = 10000000;
__global__ void test_Kernel(int* d_lrval_index_u_size,int* d_queryStream)
{
    int threadID = threadIdx.x;
    d_queryStream[threadID] = 666;
    
}

void loadQuery(string dir, std::vector<std::vector<int>>& queryStream,int &line)
{
	int r, lval, rval;
	string queryFile = dir + "querystream.txt";
	FILE * queryVec = fopen(queryFile.c_str(), "r");
	line = 0;
	while ((r = fscanf(queryVec, "%d %d", &lval, &rval)) != EOF)
	{
		if (r != 2)
		{
			fprintf(stderr, "Bad file format: u v incorrect\n");
			exit(1);
		}
		queryStream[line].resize(2);
		queryStream[line][0] = lval;
		queryStream[line][1] = rval;
		line++;
	}
	// cout<<"line: " << line;

	fclose(queryVec);
}

void cuda_query(string dir, int num_blocks_per_grid, int num_threads_per_block, int* queryAns) {
	BiGraph h_g(dir);
	lrIndexBasic(h_g);    
    vector<vector<lrval_index_block*>> h_lrval_index_u; vector<vector<lrval_index_block*>> h_lrval_index_v;
    build_lrval_index(h_g, h_lrval_index_u, h_lrval_index_v);

    size_t size = 2 * sizeof(int);
    size_t size_num_v1 = sizeof(int);
    size_t size_num_v2 = sizeof(int);




    int *d_num_v1;
    int *d_num_v2;
    
    hipMalloc((void**)&d_num_v1,size_num_v1);
    hipMalloc((void**)&d_num_v2,size_num_v2);
    hipMemcpy(d_num_v1,&h_g.num_v1,size_num_v1,hipMemcpyHostToDevice);
    hipMemcpy(d_num_v2,&h_g.num_v2,size_num_v2,hipMemcpyHostToDevice);

    // int *h_c,*d_c;
    // h_c = (int*)malloc(size);
    // hipMalloc((void**)&d_c,size);
    // hipMemcpy(d_c,h_c,size,hipMemcpyHostToDevice);
    // test_Kernel<<<num_blocks_per_grid,num_threads_per_block>>>(d_c);
    // hipMemcpy(h_c,d_c,size,hipMemcpyDeviceToHost);
    // cout<<h_c[0]<<" "<<h_c[1]<<"\n";
    // exit(0);
    int *h_lrval_index_u_size,*d_lrval_index_u_size;
    size_t size_h_lrval_index_u_size = sizeof(h_lrval_index_u.size()) * h_lrval_index_u.size();
    h_lrval_index_u_size = (int*)malloc(size_h_lrval_index_u_size);
 
    for (int i = 0;i<h_lrval_index_u.size();i++){
        h_lrval_index_u_size[i] = h_lrval_index_u[i].size();
    }
    
    hipMalloc((void**)&d_lrval_index_u_size,size_h_lrval_index_u_size);
    hipMemcpy(d_lrval_index_u_size,h_lrval_index_u_size,size_h_lrval_index_u_size,hipMemcpyHostToDevice);
    
    // test_Kernel<<<num_blocks_per_grid,num_threads_per_block>>>(d_lrval_index_u_size);


    vector<bool> left; vector<bool> right;
    // all the vertices in query result are set as true
    vector<vector<int>> queryStream;
    queryStream.resize(Q_MAX);
    int n_query = 0;
    loadQuery(dir, queryStream,n_query);
    queryStream.resize(n_query);
    cout<<n_query<<"\n";
    exit(0);
    int *h_queryStream,*d_queryStream;
    size_t size_h_query = sizeof(queryStream[0][0]) * n_query * 2
    h_queryStream = (int*)malloc(size_h_query);
    for (int i = 0;i<n_query;i++){
        h_queryStream[i*2] = queryStream[i][0];
        h_queryStream[i*2+1] = queryStream[i][1];
    }

    hipMalloc((void**)&d_queryStream,size_h_query);
    hipMemcpy(d_queryStream,h_queryStream,size_h_query,hipMemcpyHostToDevice);
    test_Kernel<<<num_blocks_per_grid,num_threads_per_block>>>(d_lrval_index_u_size,d_queryStream);
    hipMemcpy(h_queryStream,d_queryStream,size,hipMemcpyDeviceToHost);
    cout<<h_queryStream[0]<<" "<<h_queryStream[1]<<"\n";
    exit(0);


    int h_queryAns[n_query*3];
    int *d_queryAns;





}
