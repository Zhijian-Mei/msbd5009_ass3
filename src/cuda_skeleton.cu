#include "hip/hip_runtime.h"
#include "decom.h"

using namespace std;

__global__ void test_Kernel(int d_num_v1)
{
    int threadID = threadIdx.x;
    printf("threadID %-3d d_num_v1%3d\n",threadID,d_num_v1);
}

void cuda_query(string dir, int num_blocks_per_grid, int num_threads_per_block, int* queryAns) {
	BiGraph h_g(dir);
	lrIndexBasic(h_g);    
    vector<vector<lrval_index_block*>> h_lrval_index_u; vector<vector<lrval_index_block*>> h_lrval_index_v;
    build_lrval_index(h_g, h_lrval_index_u, h_lrval_index_v);

    size_t size_num_v1 sizeof(int);
    size_t size_num_v2 sizeof(int);
    size_t size_h_lrval_index_u = sizeof(lrval_index_block*)*h_lrval_index_u[0].size()*h_lrval_index_u.size();
    size_t size_h_lrval_index_v = sizeof(lrval_index_block*)*h_lrval_index_v[0].size()*h_lrval_index_v.size();
    
    int *d_num_v1;
    int *d_num_v2;
    vector<vector<lrval_index_block*>> *d_lrval_index_u;
    vector<vector<lrval_index_block*>> *d_lrval_index_v;

    hipMalloc((void**)&d_num_v1,size_num_v1);
    hipMalloc((void**)&d_num_v2,size_num_v2);
    hipMalloc(&d_lrval_index_u,size_h_lrval_index_u);
    hipMalloc(&d_lrval_index_v,size_h_lrval_index_v);
    hipMemcpy(d_num_v1,h_g.num_v1,size_num_v1,hipMemcpyHostToDevice);
    hipMemcpy(d_num_v2,h_g.num_v2,size_num_v2,hipMemcpyHostToDevice);
    hipMemcpy(d_lrval_index_u,h_lrval_index_u,size_h_lrval_index_u,hipMemcpyHostToDevice);
    hipMemcpy(d_lrval_index_v,h_lrval_index_v,size_h_lrval_index_v,hipMemcpyHostToDevice);
    test_Kernal<<<num_blocks_per_grid,num_threads_per_block>>>;
    exit(0);
    vector<bool> left; vector<bool> right;
    // all the vertices in query result are set as true
    vector<vector<int>> queryStream;
    queryStream.resize(Q_MAX);
    int n_query = 0;
    loadQuery(argv[2], queryStream, n_query);
    queryStream.resize(n_query);
    int queryAns[n_query*3];    




}
